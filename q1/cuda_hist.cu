
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define NUM_BINS 32
#define RANGE 100000

const int bin_size = RANGE / NUM_BINS;

double CLOCK() {
    struct timespec t;
    clock_gettime(CLOCK_MONOTONIC, &t);
    return (t.tv_sec * 1000) + (t.tv_nsec * 1e-6);
}

// CUDA error check macro
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error: %s (line %d)\n", hipGetErrorString(err), __LINE__); \
        exit(EXIT_FAILURE); \
    }
// Kernel function to compute histogram
__global__ void histogram_kernel(int *data, int *histogram, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int bin = data[idx] / bin_size;
        if (bin >= NUM_BINS) {
            bin = NUM_BINS - 1; // Ensure bin index is within bounds
        }

        atomicAdd(&histogram[bin], 1); // Increment the histogram bin
    }
}

int main(int argc, char** argv) {
    // Check if the number of arguments is correct
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <number_of_samples>\n", argv[0]);
        return 1;
    }

    // Parse the number of samples from the command line argument
    int num_samples = atoi(argv[1]);
    if (num_samples <= 0) {
        fprintf(stderr, "Error: Number of samples must be a positive integer.\n");
        return 1;
    }

    // number of samples in bytes
    size_t size = sizeof(int) * num_samples;

    // define host data
    int *h_data = (int *)malloc(size);
    int *h_histogram = (int *)calloc(NUM_BINS, sizeof(int)); // initialize to 0

    // initialize host data
    srand(time(NULL)); // seed in host random number generator
    for (int i = 0; i < num_samples; i++) {
        h_data[i] = rand() % RANGE + 1; // 1 - RANGE
    }

    // define device data
    int *d_data, *d_histogram;
    CUDA_CHECK(hipMalloc(&d_data, size));
    CUDA_CHECK(hipMalloc(&d_histogram, NUM_BINS * sizeof(int)));

    // copy data from host to device
    CUDA_CHECK(hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_histogram, h_histogram, NUM_BINS * sizeof(int), hipMemcpyHostToDevice));

    // Start benchmarking time
    double start, stop;
    start = CLOCK();

    // define block and grid sizes
    int threads = 512; // number of threads per block
    int blocks = (num_samples + threads - 1) / threads; // number of blocks
    histogram_kernel<<<blocks, threads>>>(d_data, d_histogram, num_samples);
    CUDA_CHECK(hipGetLastError()); // check for kernel launch errors

    // copy histogram from device to host
    CUDA_CHECK(hipMemcpy(h_histogram, d_histogram, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost));

    // Stop benchmarking time
    stop = CLOCK();

    printf("Time taken: %f ms\n", stop - start);
    printf("Num. of Samples: %d\n", num_samples);

    // print histogram
    for (int i = 0; i < NUM_BINS; i++) {
        printf("Bin %d: %d\n", i, h_histogram[i]);
    }

    // free device memory
    hipFree(d_data);
    hipFree(d_histogram);
    
    // free host memory
    free(h_data);
    free(h_histogram);

    return 0;
}   

